#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "include/velectric.hpp"

#define K 8.99E9

__host__ __device__ inline double square(double x) {
    return x * x;
}

__host__ __device__ inline double Vec2::magnitude() const {
    return sqrt(square(this->x) + square(this->y));
}

__host__ __device__ inline Vec2 Vec2::unitVector() const {
    const double mag = this->magnitude();
    return {this->x/mag, this->y/mag};
}

__device__ inline Vec2 electricField(double charge, Vec2 r) {
    Vec2 eField = {0, 0};
    const double magn = r.magnitude();
    Vec2 unitR = r.unitVector();
    eField.x = (K * charge / square(magn)) * unitR.x;
    eField.y = (K * charge / square(magn)) * unitR.y;
    return eField;  
}

void Scene::compute() {
    // variable for current positions of charges, set to intial value
    // variable for current vel
    // variable for current accel
    // above variables must be in scene space with integer coordinates
    // For each timestep:
    //      Compute electric field due to each point charge at their position (cuda)
    //      add individual electric fields  (cuda)
    //      for each charge (cuda):
    //              determine acceleration using field, charge, mass
    //              determine velocity using previous velocity, previous acceleration, previous position, and time differential
    //              determine position using previous velocity, previous acceleration, previous position, and time differential
    //              update all variables and additionally add to positions list.
}

__global__ void addFields(Vec2* sum, Vec2* field1, Vec2* field2) {
    const int i = threadIdx.x + blockIdx.x * blockDim.x;  // i is guaranteed to be between 0 and the capacity due to the threads we allocate
    sum[i].x = field1[i].x + field2[i].x;
    sum[i].y = field1[i].y + field2[i].y;
}

__global__ void computeField(Vec2* field, Vec2 pos, double rows, double cols, Vec2 bottomRight, Vec2 bottomLeft, double charge) {
    /* charge is at row pos[0], column pos[1]. */
    
}

