#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <velectric.hpp>
#include <velectric.cuh>

#define K 8.99E9

__host__ __device__ inline double square(double x) {
    return x * x;
}

// Vec2
__device__ __host__ Vec2 Vec2::operator*(double a) const {
    return {x * a, y * a};
}

__device__ __host__ Vec2 Vec2::operator+(const Vec2 &other) const {
    return {x + other.x, y + other.y};
}

__device__ __host__ Vec2 Vec2::operator-(const Vec2 &final) const {
    return {*this + (final)*(-1)};
}

__device__ __host__ void Vec2::operator*=(double a){
    *this = *this * a;
}

__host__ __device__ inline double Vec2::magnitude() const {
    return sqrt(square(this->x) + square(this->y));
}

__host__ __device__ inline Vec2 Vec2::unitVector() const {
    const double mag = this->magnitude();
    return {this->x/mag, this->y/mag};
}
// Vec2

__device__ inline Vec2 electricField(double charge, Vec2 r) {
    Vec2 eField = {0, 0};
    const double magn = r.magnitude();
    Vec2 unitR = r.unitVector();
    eField.x = (K * charge / magn*magn) * unitR.x;
    eField.y = (K * charge / magn*magn) * unitR.y;
    return eField;  
}

void Scene::compute() {
    // variable for current positions of charges, set to intial value
    // variable for current vel
    // variable for current accel
    // above variables must be in scene space with integer coordinates
    // For each timestep:
    //      Compute electric field due to each point charge at their position (cuda)
    //      add individual electric fields  (cuda)
    //      for each charge (cuda):
    //              determine acceleration using field, charge, mass
    //              determine velocity using previous velocity, previous acceleration, previous position, and time differential
    //              determine position using previous velocity, previous acceleration, previous position, and time differential
    //              update all variables and additionally add to positions list.
}

__global__ void addFields(Vec2* sum, Vec2* field1, Vec2* field2) {
    const int i = threadIdx.x + blockIdx.x * blockDim.x;  // i is guaranteed to be between 0 and the capacity due to the threads we allocate
    sum[i].x = field1[i].x + field2[i].x;
    sum[i].y = field1[i].y + field2[i].y;
}

__global__ void computeField(Vec2* &field, Vec2 bottomLeft, double charge, size_t chargeRow, size_t chargeCol, double dx, size_t length, size_t rows) {
    // computes the electric field at every point in scene space, but stores those values in real space
    const size_t rowLen = length/rows;
    const size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    // don't do anything if at the same point as the field
    if (i == rowLen*(chargeRow)+chargeCol) {
        return;
    }
    const size_t row = i/rowLen;
    const size_t col = i%rowLen;
    Vec2 op = {bottomLeft.x + col*dx, bottomLeft.y + row*dx};
    Vec2 oq = {bottomLeft.x + chargeCol*dx, bottomLeft.y + chargeRow*dx};
    field[i] = electricField(charge, op - oq);
}

