#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "include/velectric.hpp"

void Scene::compute() {
    // variable for current positions of charges, set to intial value
    // variable for current vel
    // variable for current accel
    // above variables must be in scene space with integer coordinates
    // For each timestep:
    //      Compute electric field due to each point charge at their position (cuda)
    //      add individual electric fields  (cuda)
    //      for each charge (cuda):
    //              determine acceleration using field, charge, mass
    //              determine velocity using previous velocity, previous acceleration, previous position, and time differential
    //              determine position using previous velocity, previous acceleration, previous position, and time differential
    //              update all variables and additionally add to positions list.
}

__global__ void addFields(std::array<double, 2>* sum, std::array<double, 2>* field1, std::array<double, 2>* field2) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;  // i is guaranteed to be between 0 and the capacity due to the threads we allocate
    sum[i][0] = field1[i][0] + field2[i][0];
    sum[i][1] = field1[i][1] + field2[i][1];
}

