#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "include/velectric.hpp"

#define K 8.99E9

__device__ inline double square(double x) {
    return x * x;
}

__device__ inline double magnitude(std::array<double, 2> v) {
    return sqrt(square(v[0]) + square(v[1]));
}

__device__ inline std::array<double, 2> unitVector(std::array<double, 2> v) {
    return {v[0]/magnitude(v), v[1]/magnitude(v)};
}

__device__ inline std::array<double, 2> electricField(double charge, std::array<double, 2> r) {
    std::array<double, 2> eField = {0, 0};
    double magn = magnitude(r);
    std::array<double, 2> unitR = unitVector(r);
    eField[0] = (K * charge / square(magn)) * unitR[0];
    eField[1] = (K * charge / square(magn)) * unitR[1];
    return eField;
}

void Scene::compute() {
    // variable for current positions of charges, set to intial value
    // variable for current vel
    // variable for current accel
    // above variables must be in scene space with integer coordinates
    // For each timestep:
    //      Compute electric field due to each point charge at their position (cuda)
    //      add individual electric fields  (cuda)
    //      for each charge (cuda):
    //              determine acceleration using field, charge, mass
    //              determine velocity using previous velocity, previous acceleration, previous position, and time differential
    //              determine position using previous velocity, previous acceleration, previous position, and time differential
    //              update all variables and additionally add to positions list.
}

__global__ void addFields(std::array<double, 2>* sum, std::array<double, 2>* field1, std::array<double, 2>* field2) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;  // i is guaranteed to be between 0 and the capacity due to the threads we allocate
    sum[i][0] = field1[i][0] + field2[i][0];
    sum[i][1] = field1[i][1] + field2[i][1];
}

__global__ void computeField(std::array<double, 2>* field, std::array<size_t, 2> position) {

}

